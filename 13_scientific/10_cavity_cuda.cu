
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>

#define nx 41
#define ny 41

#define N_THREAD 1024
#define N_BLOCK (ny * nx + N_THREAD - 1) / N_THREAD

using namespace std;

__device__ float u[ny][nx];
__device__ float v[ny][nx];
__device__ float p[ny][nx];
__device__ float b[ny][nx];
__device__ float un[ny][nx];
__device__ float vn[ny][nx];
__device__ float pn[ny][nx];

__device__ double dx = 2. / (nx - 1);
__device__ double dy = 2. / (ny - 1);
__device__ double dt = .01;
__device__ double rho = 1.;
__device__ double nu = .02;

__global__ void init() {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int j = id / nx;
  int i = id % nx;
  if (j >= ny) return;

  u[j][i] = 0;
  v[j][i] = 0;
  p[j][i] = 0;
  b[j][i] = 0;
}

__global__ void compute_b() {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int j = id / nx;
  int i = id % nx;
  if (j >= ny) return;
  
  // Compute b[j][i]
  float dudx = (u[j][i+1] - u[j][i-1]) / (2 * dx);
  float dudy = (u[j][i+1] - u[j][i-1]) / (2 * dy);
  float dvdx = (v[j+1][i] - v[j-1][i]) / (2 * dx);
  float dvdy = (v[j+1][i] - v[j-1][i]) / (2 * dy);
  b[j][i] = rho * (1 / dt * (dudx + dvdy)
		 - dudx * dudx - 2 * dudy * dvdx - dvdy * dvdy);
}   

__global__ void copy_p() {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int j = id / nx;
  int i = id % nx;
  if (j >= ny) return;
  
  pn[j][i] = p[j][i];
}

__global__ void compute_p() {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int j = id / nx;
  int i = id % nx;
  if (j >= ny) return;

  float *a = &p[j][i];

  if (j == 0) j = 1;
  else if (j == ny-1) { *a = 0; return; }
  else if (i == 0) i = 1;
  else if (i == nx-1) i = nx-2;
  
  // Compute p[j][i]
  *a = (dy*dy * (pn[j][i+1] + pn[j][i-1]) +
             dx*dx * (pn[j+1][i] + pn[j-1][i]) -
             b[j][i] * dx*dx * dy*dy)
           / (2 * (dx*dx + dy*dy));
}

__global__ void copy_uv() {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int j = id / nx;
  int i = id % nx;
  if (j >= ny) return;

  un[j][i] = u[j][i];
  vn[j][i] = v[j][i];
}

__global__ void compute_uv() {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int j = id / nx;
  int i = id % nx;
  if (j >= ny) return;
  
  // Compute u[j][i] and v[j][i]
  if (j == 0) {
    u[0][i] = 0;
    v[0][i] = 0;
    return;
  }
  if (j == ny-1) {
    u[ny-1][i] = 1;
    v[ny-1][i] = 0;
    return;
  }
  if (i == 0 || i == nx-1) {
    u[j][i] = 0;
    v[j][i] = 0;
    return;
  }
  
  u[j][i] = un[j][i] - un[j][i] * dt / dx * (un[j][i] - un[j][i - 1])
                     - vn[j][i] * dt / dy * (un[j][i] - un[j - 1][i])
                     - dt / (2 * rho * dx) * (p[j][i+1] - p[j][i-1])
                     + nu * dt / (dx*dx) * (un[j][i+1] - 2 * un[j][i] + un[j][i-1])
                     + nu * dt / (dy*dy) * (un[j+1][i] - 2 * un[j][i] + un[j-1][i]);
  v[j][i] = vn[j][i] - un[j][i] * dt / dx * (vn[j][i] - vn[j][i - 1])
                     - vn[j][i] * dt / dy * (vn[j][i] - vn[j - 1][i])
                     - dt / (2 * rho * dx) * (p[j+1][i] - p[j-1][i])
                     + nu * dt / (dx*dx) * (vn[j][i+1] - 2 * vn[j][i] + vn[j][i-1])
                     + nu * dt / (dy*dy) * (vn[j+1][i] - 2 * vn[j][i] + vn[j-1][i]);
}

float u_cpu[ny][nx];
float v_cpu[ny][nx];
float p_cpu[ny][nx];

int main() {
  int nt = 500;
  int nit = 50;

  init<<<N_BLOCK, N_THREAD>>>();
  hipDeviceSynchronize();

  ofstream ufile("u.dat");
  ofstream vfile("v.dat");
  ofstream pfile("p.dat");
  for (int n=0; n<nt; n++) {
    compute_b<<<N_BLOCK, N_THREAD>>>();
    hipDeviceSynchronize();
    for (int it=0; it<nit; it++) {
      copy_p<<<N_BLOCK, N_THREAD>>>();
      hipDeviceSynchronize();

      compute_p<<<N_BLOCK, N_THREAD>>>();
      hipDeviceSynchronize();
    }
    copy_uv<<<N_BLOCK, N_THREAD>>>();
    hipDeviceSynchronize();

    compute_uv<<<N_BLOCK, N_THREAD>>>();
    hipDeviceSynchronize();

    if (n % 10 == 0) {
      hipMemcpyFromSymbol(u_cpu, HIP_SYMBOL(u), sizeof(float) * ny * nx, 0);
      hipMemcpyFromSymbol(v_cpu, HIP_SYMBOL(v), sizeof(float) * ny * nx, 0);
      hipMemcpyFromSymbol(p_cpu, HIP_SYMBOL(p), sizeof(float) * ny * nx, 0);

      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          ufile << u_cpu[j][i] << " ";
      ufile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          vfile << v_cpu[j][i] << " ";
      vfile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          pfile << p_cpu[j][i] << " ";
      pfile << "\n";
    }
  }
  ufile.close();
  vfile.close();
  pfile.close();
}


